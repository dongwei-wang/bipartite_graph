
#include <hip/hip_runtime.h>
#include <iostream>
#include <utility>
#include <algorithm>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <time.h>
//#include "graphfilter.h"

//#include <cutil.h>

//#include <graphfilter_kernel.cu>
#define N 400
#define DEBUG 0

typedef struct _Graph_node_A {
	int adj[N];
} Graph_node_A;

using namespace std;

extern "C"

#ifndef _MATRIXMUL_H_
#define _MATRIXMUL_H_
__global__ void naiveFilterKernel(struct _Graph_node_A *PA, int *relation_count, int *v_start, int *max_id_A)
{
	__shared__ int v_start_adj[N];

	int tid = threadIdx.x;
	int count=0;

	if(tid < N)
		v_start_adj[tid] = PA[(*v_start)-1].adj[tid];
	__syncthreads();

	if(tid == ((*v_start)-1) || tid >= *max_id_A) return;

	int i, j;

	for(i=0;(i<N) && (PA[tid].adj[i] != 0);i++) {
		for(j=0;j<N && v_start_adj[j]!= 0;j++) {
			if(PA[tid].adj[i] == v_start_adj[j]){
				count++;
				break;
			}
		}
	}
	relation_count[tid] = count;
}
#endif

#ifndef _MATRIXMUL_F_
#define _MATRIXMUL_F_
__global__ void fullFilterKernel(struct _Graph_node_A *PA, int *relation_count, int *index_vertice, int *k, int *max_id_A)
{
	__shared__ int index_ref[N];

	int by = blockIdx.y;
	int tx = threadIdx.x;
	int count = 0;

	if((by >= *k) || (tx >= *max_id_A)) return;

	// load data into shared memory
	if(tx < N)
		index_ref[tx] = PA[index_vertice[by]-1].adj[tx];
	__syncthreads();

	for(int i=0;(i<N) && (PA[tx].adj[i] != 0);i++)
		for(int j=0;(j<N) && (index_ref[j]!= 0);j++) {
			if((PA[tx].adj[i] == index_ref[j]) && ((index_vertice[by]-1) != tx)) {
				count++;
				break;
			}
		}

	relation_count[(by+1) * (*max_id_A) + tx] = count;
}
#endif

void filterOnDevice(struct _Graph_node_A *PA, int *relation_count, int *index_vertice, int v_start, int k, int max_id_A);
void filterOnHost(struct _Graph_node_A *PA, int *relation_count_h, int *index_vertice_h, int v_start, int k, int max_id_A);

// function to sort the relations to the start vertice in a descending order
void selSort(int s[], int index[], int length);

//function to compare the result between the host side and device side
bool compare(vector<int> final_resut_h, vector<int> final_result_d);

//static const char* DAT_FILE_NAME = "graph.dat";// file to store the graph struct

int main(int arcg, char** argv)
{
	char tracefilename[30];
	int A, B; //fot the nodes in party A and party B
	int max_id_A;
	int max_id_B;
	int max_weight;
	int i=0, j=0;
	//int m=0, n=0;
	FILE *fp;
	char buf[100];

	struct _Graph_node_A *PA;

	/***********************************************************
	  read in the trace file **********************************
	 ************************************************************/
	printf("Please input the trace file name:");
	scanf("%s", &tracefilename);

	fp = fopen(tracefilename, "r");
	if(fp==NULL){
		printf("Could not open the trace file!\n");
		exit(1);
	}
	printf("Has open the trace successfully!\n");

	while(fgets(buf,100,fp)){
		if(buf[0]=='%') continue;
		if(i==0){
			sscanf(buf, "%d%d%d", &max_id_A, &max_id_B, &max_weight);
			break;
		}
	}

	fclose(fp);

	cout << max_id_A << endl;
	PA = (struct _Graph_node_A *)malloc(sizeof(struct _Graph_node_A) * max_id_A);
	if(PA == NULL)
		cout << "Allocate memory for PA failed" <<endl;
	else
		cout << "Allocate memory for PA successfully" << endl;

	// initialize A and B
	for(i=0;i<max_id_A;i++)
		for(j=0;j<N;j++)
			PA[i].adj[j] = 0;

	cout << "Initialized PA successfully!" << endl;

	fp = fopen(tracefilename, "r");
	if(fp==NULL){
		printf("Could not open the trace file!\n");
		exit(1);
	}
	printf("Has open the trace sucessfully!\n");

	int index[max_id_A];
	for(i=0;i<max_id_A;i++)
		index[i]  = 0;
	cout << "Initilized index successfully" << endl;

	i=0;

	//read in the input file and build the graph
	while(fgets(buf,100,fp)){
		if(buf[0]=='%')
			continue;
		if(i==0){
			sscanf(buf,"%d%d%d", &max_id_A, &max_id_B, &max_weight);
			cout << max_id_A << " " << max_id_B << " " << max_weight << endl;
			i++;
		}else{
			sscanf(buf, "%d%d", &A, &B);
			PA[A-1].adj[index[A-1]++] = B;
		}
	}

	fclose(fp);

	// Naive Graph collaboratie Filtering
	int v_start;// the start vertive for search
	int k;// the k is the number of relational vertices needed to be found

	// read in the start vertex, and value k for ralation depth
	std::cout << "Please input the start vertice as an interger number less than " << max_id_A << " :";
	scanf("%d", &v_start);
	while(v_start<0 || v_start > max_id_A) {
		cout << "Please input a valid start vertice less than " << max_id_A << " :";
		scanf("%d", &v_start);
	}

	std::cout << "Please input the value of k as an interger number less than " << max_id_A << " :";
	scanf("%d", &k);
	while(k < 0 || k > max_id_A) {
		cout << "Please input a valid k less than " << max_id_A << " :";
		scanf("%d", &k);
	}


	int* relation_count = (int*)malloc((k+1)*max_id_A*sizeof(int));
	int* index_vertice = (int*)malloc((k+1)*max_id_A*sizeof(int));

	int* relation_count_h = (int*)malloc((k+1)*max_id_A*sizeof(int));
	int* index_vertice_h = (int*)malloc((k+1)*max_id_A*sizeof(int));

	//initialize the relation_count and vertice index
	for(i=0;i<k+1;i++)
		for(j=0;j<max_id_A;j++){
			relation_count[i * max_id_A + j] = 0;
			relation_count_h[i * max_id_A + j] = 0;
			index_vertice[i * max_id_A + j] = j + 1;
			index_vertice_h[i * max_id_A + j] = j + 1;
		}

	// executing the filtering algorithm on the host side
	clock_t st = clock();
	filterOnHost(PA, relation_count_h, index_vertice_h, v_start, k, max_id_A);
	st = clock() - st;
	printf("CPU execution time is %.5f\n", (float)st/CLOCKS_PER_SEC);

#if DEBUG
	cout << "Relation count on the host side is:" << endl;
	for(i=0;i<k+1;i++){
		for(j=0;j<max_id_A;j++){
			cout << relation_count_h[i*max_id_A+j] << " ";
		}
		cout << endl;
	}
#endif

	// executing the filtering algorithm on the device side
	st = clock();
	filterOnDevice(PA, relation_count, index_vertice, v_start, k, max_id_A);
	st = clock() - st;
	printf("GPU execution time is %.5f\n", (float)st/CLOCKS_PER_SEC);


	//sort the relation in a descending order
	for(i=1;i<k+1;i++) {
		selSort(&relation_count[i*max_id_A], &index_vertice[i*max_id_A], max_id_A);
		selSort(&relation_count_h[i*max_id_A], &index_vertice_h[i*max_id_A], max_id_A);
	}

#if DEBUG
	cout << "The " << k << " related vertices to each vertice are: " << endl;
	for(i=0;i<k+1;i++) {
		for(j=0;j<k;j++) {
			cout << index_vertice[i * max_id_A + j]	<< " ";
			cout << index_vertice_h[i * max_id_A + j] << " ";
		}
		cout << endl;
	}
#endif


	vector<int> final_result_d;// record the final result of collaborative filtering for visualization
	vector<int> final_result_h;
	vector<int>::iterator it;

	// copy the index_vertice to the final_result vector
	for(i=0;i<k+1;i++)
		for(j=0;j<k;j++) {
			final_result_d.push_back(index_vertice[i * max_id_A + j]);
			final_result_h.push_back(index_vertice_h[i * max_id_A + j]);
		}


	// sort the final_result vector in a desending order
	std::sort(final_result_d.begin(), final_result_d.end());
	std::sort(final_result_h.begin(), final_result_h.end());

	// remove the repeated vertices
	for(it=final_result_d.begin()+1, i=final_result_d.front();it!=final_result_d.end();) {
		if(i == *it)
			final_result_d.erase(it);
		else {
			i = *it;
			it++;
		}
	}

	for(it=final_result_h.begin()+1, i=final_result_h.front();it!=final_result_h.end();) {
		if(i == *it)
			final_result_h.erase(it);
		else {
			i = *it;
			it++;
		}
	}

	// compare the result from GPU with the result from CPU to test the correctness
	bool match = compare(final_result_h, final_result_d);
	if(match)
		cout << "Test passed ^^!" << endl;
	else
		cout << "Test failed !!" << endl;

	// output the final result
	cout << "The final Collaborative Filtering result is:" << endl;
	for(it=final_result_d.begin();it!=final_result_d.end();it++)
		cout << *it << " ";
	cout << endl;

	free(relation_count);
	free(relation_count_h);
	free(index_vertice);
	free(index_vertice_h);

	return 0;
}

void filterOnHost(struct _Graph_node_A *PA, int *relation_count, int *index_vertice, int v_start, int k, int max_id_A)
{
	int i, j, m, n;

	// naive collaborative filtering
	for(i=0;i<max_id_A;i++){
		if(i == v_start - 1)	continue;
		for(j=0;j<N && PA[i].adj[j] != 0;j++)
			for(m=0;m<N && PA[v_start-1].adj[m] != 0;m++){
				if(PA[v_start-1].adj[m] == PA[i].adj[j]) {
					relation_count[i]++;
					break;
				}
			}
	}

	// sort the relation in a descending order by selection sort algorithm
	selSort(relation_count, index_vertice, max_id_A);

	// full collaborative filtering
	for(i=0;i<k;i++)
		for(j=0;j<max_id_A;j++)
			for(m=0;m<N && PA[j].adj[m] != 0;m++)
				for(n=0;n<N && PA[index_vertice[i]-1].adj[n] != 0;n++) {
					if(PA[index_vertice[i]-1].adj[n] == PA[j].adj[m] && index_vertice[i] - 1 != j) {
						relation_count[(i+1)*max_id_A + j] ++;
						break;
					}
				}

}

void filterOnDevice(struct _Graph_node_A *PA, int *relation_count, int *index_vertice, int v_start, int k, int max_id_A) {
	struct _Graph_node_A *PA_d;
	int *relation_count_d;
	int *max_id_A_d;
	int *index_vertice_d;
	int *v_start_d;
	int *k_d;
	int size = (k + 1) * max_id_A * sizeof(int);

	hipMalloc((void**)&(PA_d), sizeof(struct _Graph_node_A) * max_id_A);
	hipMalloc((void**)&(relation_count_d), size);
	hipMalloc((void**)&(max_id_A_d), sizeof(int));
	hipMalloc((void**)&(v_start_d), sizeof(int));
	hipMalloc((void**)&(k_d), sizeof(int));
	hipMalloc((void**)&(index_vertice_d), max_id_A * sizeof(int));

	hipMemcpy(PA_d, PA, sizeof(struct _Graph_node_A) * max_id_A, hipMemcpyHostToDevice);
	hipMemcpy(relation_count_d, relation_count, size, hipMemcpyHostToDevice);
	hipMemcpy(max_id_A_d, &max_id_A, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(v_start_d, &v_start, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(k_d, &k, sizeof(int), hipMemcpyHostToDevice);

	// naive collaborative filtering
	dim3 dimGrid0(1, 1);
	dim3 dimBlock0(max_id_A, 1);
	naiveFilterKernel<<<dimGrid0, dimBlock0>>>(PA_d, relation_count_d, v_start_d, max_id_A_d);

	hipMemcpy(relation_count, relation_count_d, max_id_A * sizeof(int), hipMemcpyDeviceToHost);

#if DEBUG
	cout << "The relation count is:" << endl;
	for(int i=0; i<k+1;i++) {
		for(int j=0;j<max_id_A;j++){
			cout << relation_count[i*max_id_A+j] << " ";
		}
		cout << endl;
	}
#endif

	selSort(relation_count, index_vertice, max_id_A);

#if DEBUG
	cout << "The " << k << " related vertice to " << v_start << " is:" << endl;
	for(int i=0;i<k;i++)
		cout << index_vertice[i] << " ";
	cout << endl;
#endif

	hipMemcpy(index_vertice_d, index_vertice, max_id_A * sizeof(int), hipMemcpyHostToDevice);

	// full collaborative filtering
	dim3 dimGrid(1, k);
	dim3 dimBlock(max_id_A, 1);
	fullFilterKernel<<<dimGrid, dimBlock>>>(PA_d, relation_count_d, index_vertice_d, k_d, max_id_A_d);

	hipMemcpy(relation_count, relation_count_d, size, hipMemcpyDeviceToHost);

#if DEBUG
	cout << "The relation count is:" << endl;
	for(int i=0; i<k+1;i++) {
		for(int j=0;j<max_id_A;j++){
			cout << relation_count[i*max_id_A+j] << " ";
		}
		cout << endl;
	}
#endif

	hipFree(PA_d);
	PA_d = NULL;
	hipFree(relation_count_d);
	relation_count_d = NULL;
	hipFree(max_id_A_d);
	max_id_A_d = NULL;
	hipFree(k_d);
	k_d = NULL;
	hipFree(index_vertice_d);
	index_vertice_d = NULL;
}

// Selection Sort
void selSort(int s[], int index[], int length)
{
	int i, j, maxPos;
	for(i=0;i<length-1;i++) {
		maxPos = i;
		for(j=i+1;j<length;j++)
			if(s[j] > s[maxPos])
				maxPos = j;
		if(i != maxPos) {
			swap(s[i], s[maxPos]);
			swap(index[i], index[maxPos]);
		}
	}
}

bool compare(vector<int> final_result_h, vector<int> final_result_d)
{
	vector<int>::iterator it_h;
	vector<int>::iterator it_d;

	for(it_h=final_result_h.begin(), it_d=final_result_d.begin(); it_h != final_result_h.end() && it_d != final_result_d.end();it_h++, it_d++) {
		if((*it_h) != (*it_d))
			return false;
	}

	return true;
}
